#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifndef ALIGNMENT
#define ALIGNMENT 64
#endif
#include <stdio.h>
#include <string>
#include <map>
#include <iostream>
#include <fstream>
#include <chrono>

using namespace std;

#define O (2048 * 2048)
#define THREADS_PER_BLOCK 512
__global__ void add(int *a, int *b, int *c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}
__global__ void dot(int*a, int*b, int*c)
{
	__shared__ int temp[THREADS_PER_BLOCK];
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	temp[threadIdx.x] = a[index] * b[index];

	__syncthreads();

	if (0 == threadIdx.x)
	{
		int sum = 0;
		for (int i = 0; i< THREADS_PER_BLOCK; i++)
			sum += temp[i];
		atomicAdd(c, sum);
	}
}

void random_ints(int* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
		a[i] = rand();
}

bool singleCore = true;

__global__ void fwStepK(int k, int devArray[], int Na, int N) 
{
	int col = blockIdx.x * blockDim.x + threadIdx.x; 
	/* This thread's matrix column */
	if (col >= N)
		return;
	int arrayIndex = Na * blockIdx.y + col; __shared__ int trkc;
	/* this row, kth column */
	// Improve by using the intermediate k, if we can
	if(threadIdx.x == 0)
		trkc = devArray[Na * blockIdx.y + k];
	__syncthreads();
	if(trkc == INT_MAX)
		/* infinity */
		return;
	int tckr = devArray[k*Na + col]; 
	/* this column, kth row */
	if(tckr == INT_MAX)    
		/* infinity */
		return;
	int betterMaybe = trkc + tckr;
	if(betterMaybe < devArray[arrayIndex])
		devArray[arrayIndex] =  betterMaybe;
}
map<string, int> nameToNum;
/* names of vertices */
map<string, map<string, int>> weightMap;
/* weights of edges */
int* readGraph(int& N, int& Na, char* argv)
{
	// Read the graph file from memory
	string vname1, vname2;
	ifstream graphFile;
	string dummyString;
	int thisWeight;
	/* weight of the edge just read from file */
	N = 0;
	/* number of vertices */
	graphFile.open(argv);
	//Read the graph into some maps
	graphFile >> vname1;
	while (vname1 != "--END--")
	{
		graphFile >> vname2; graphFile >> thisWeight;
		if (nameToNum.count(vname1) == 0)
		{
			nameToNum[vname1] = N;
			weightMap[vname1][vname1] = 0;
			N++;
		}
		if (nameToNum.count(vname2) == 0)
		{
			nameToNum[vname2] = N; weightMap[vname2][vname2] = 0;
			N++;
		}
		weightMap[vname1][vname2] = thisWeight;
		graphFile >> vname1;
	}
	graphFile.close(); // Nice and Tidy// "alignment" is what stored row sizes must be a multiple of30
	int alignment = ALIGNMENT;
	if (!singleCore)
	{
		Na = alignment*((N + alignment - 1) / alignment);
	}
	else
	{
		Na = N;
	}
	/* for the sizes of our arrays */
	printf("Alignment = %d\n", alignment);
	// Build the array
	int* a = (int*)malloc(N*Na * sizeof(int));
	for (int ii = 0; ii < N; ii++)
		for (int jj = 0; jj < N; jj++)
			a[ii * Na + jj] = INT_MAX;
	map<string, int>::iterator i;
	map<string, int>::iterator j;
	for (i = nameToNum.begin(); i != nameToNum.end(); ++i)
		for (j = nameToNum.begin(); j != nameToNum.end(); ++j)
		{
			if (weightMap[(*i).first].count((*j).first) != 0)
			{
				a[Na * (*i).second + (*j).second] = weightMap[(*i).first][(*j).first];
			}
		}
	return a;
}
void printArray(int Na, int* a)
{
	map<string, int>::iterator i, j;
	for (i = nameToNum.begin(); i != nameToNum.end(); ++i)
		if (i->second < 10)
			printf("\t%s", i->first.c_str());
	printf("\n"); j = nameToNum.begin();
	for (i = nameToNum.begin(); i != nameToNum.end(); ++i)
	{
		if (i->second < 10)
		{
			printf("%s\t", i->first.c_str());
			for (j = nameToNum.begin(); j != nameToNum.end(); ++j)
			{
				if (j->second < 10)
				{
					int dd = a[i->second * Na + j->second];
					if (dd != INT_MAX)
						printf("%d\t", dd);
					else
						printf("--\t");
				}
			}
		}
	}
}
int main(int argc, char* argv[])
{
	std::chrono::time_point<std::chrono::steady_clock> begin = std::chrono::high_resolution_clock::now();
	/*int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = O * sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, sizeof(int));

	a = (int *)malloc(size); random_ints(a, O);
	b = (int *)malloc(size); random_ints(b, O);
	c = (int *)malloc(sizeof(int));

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	std::chrono::time_point<std::chrono::steady_clock> mid1 = std::chrono::high_resolution_clock::now();
	dot<<< O/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

	std::chrono::time_point<std::chrono::steady_clock> mid2 = std::chrono::high_resolution_clock::now();

	hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	std::chrono::time_point<std::chrono::steady_clock> mid3 = std::chrono::high_resolution_clock::now();
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);*/

	if (!singleCore)
	{
		int N = 0;
		int Na = 0;
		// Width of matrix to encourage coalescing
		int* graph = readGraph(N, Na, "graph.txt");
		// from fwHelpers.cpp
		printf("Kernel: Just read %s with %d vertices, Na = %d\n", argv[1], N, Na);
		// Copy the array to newly-allocated global memory
		int* devArray;
		hipError_t err = hipMalloc(&devArray, Na*N * sizeof(int));
		printf("Malloc device rules: %s\n", hipGetErrorString(err));
		err = hipMemcpy(devArray, graph, Na*N * sizeof(int), hipMemcpyHostToDevice);
		printf("Pre-kernel copy memory onto device: %s\n", hipGetErrorString(err));
		// Set up and run the kernels
		int threadsPerBlock = 256;
		dim3 blocksPerGrid((Na + threadsPerBlock - 1) / threadsPerBlock, N);
		// The kth run through this loop considers whether we might do better using// the kth vertex as an intermediate
		for (int k = 0; k < N; k++)
		{
			fwStepK <<< blocksPerGrid, threadsPerBlock >>> (k, devArray, Na, N);
			err = hipDeviceSynchronize();
			// Uncomment the following line when debugging the kernel
			// printf("Kernel: using %d as intermediate: error = %s\n", k, hipGetErrorString(err));
			// Uncomment the following two lines to print intermediate results
			// err = hipMemcpy(graph, devArray, Na*N*sizeof(int), hipMemcpyDeviceToHost);
			// printArray(Na, graph);
		}
		err = hipMemcpy(graph, devArray, Na*N * sizeof(int), hipMemcpyDeviceToHost);
		printf("Post-kernel copy memory off of device: %s\n", hipGetErrorString(err));
		printArray(Na, graph);
		free(graph);
		hipFree(devArray);
	}
	else
	{
		int N = 0; 
		int Na = 0; 
		int* graph = readGraph(N, Na, "graph.txt");
		printf("Read %s with %d vertices, Na = %d\n", argv[1], N, Na); 
		printArray(N, graph); 
		for (int k = 0; k < N; k++)
			for (int i = 0; i < N; i++)
				if (graph[i*N + k] != INT_MAX)
					for (int j = 0; j < N; j++)
						if (graph[k*N + j] != INT_MAX)
							if (graph[i*N + k] + graph[k*N + j] < graph[i*N + j])
								graph[i*N + j] = graph[i*N + k] + graph[k*N + j]; 
		printArray(N, graph);
	}
	
	std::chrono::time_point<std::chrono::steady_clock> end = std::chrono::high_resolution_clock::now(); 

	/*float percent1 = 1.0f * std::chrono::duration_cast<std::chrono::nanoseconds>(mid1 - begin).count() / std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
	float percent2 = 1.0f * std::chrono::duration_cast<std::chrono::nanoseconds>(mid3 - mid2).count() / std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
	std::cout << std::endl << percent1 << std::endl;
	std::cout << std::endl << percent2 << std::endl;*/
	std::cout << std::endl << std::endl << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() << "ns" << std::endl;

	int z;
	scanf("%d", &z);

    return 0;
}